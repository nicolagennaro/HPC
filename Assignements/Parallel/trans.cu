#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 64

typedef double MYTYPE;


__global__ void mat_trans(MYTYPE* a, MYTYPE* at, int size){
  // MYTYPE tmp;

  // threadIdx from 0 to NUM_THREADS
  // blockIdx = (size*size + NUM_THREADS)/NUM_THREADS
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row = idx/size;
  int col = idx % size;

  if( idx < size*size ){
    at[ col*size + row ] = a[ row*size + col ];
  }
}



void mat_print(MYTYPE *m, int dim){
  int i, j;
  printf("\n");
  for(i=0; i<dim; i++){
	 for(j=0; j<dim; j++)
	 	  printf( "%.3lf\t", m[i*dim + j] );
	printf("\n");
  }
  printf("\n");
}




int main(int argc, char *argv[]){

MYTYPE *h_a, *h_at;
MYTYPE *d_a, *d_at;
int dim, i;
size_t m_size;

if( argc != 2){
 printf("\n");
 printf("usage: a.out MAT_DIM\n\n");
 exit(1);
}

dim = atoi(argv[1]);
m_size = dim*dim * sizeof(MYTYPE);

h_a = (MYTYPE*) malloc( m_size );
h_at = (MYTYPE*) malloc( m_size );

for(i=0; i<dim*dim; i++)
	 h_a[i] = (MYTYPE)( rand() % 10 );

if( dim <= 10)
    mat_print(h_a, dim);

hipMalloc((void**) &d_a, m_size );
hipMalloc((void**) &d_at, m_size );


hipMemcpy(d_a, h_a, m_size, hipMemcpyHostToDevice);

mat_trans<<< (dim*dim + NUM_THREADS)/NUM_THREADS , NUM_THREADS >>>(d_a, d_at, dim);

hipMemcpy(h_at, d_at, m_size, hipMemcpyDeviceToHost);


if( dim <= 10)
    mat_print(h_at, dim);

free(h_a);
free(h_at);

hipFree(d_a);
hipFree(d_at);

return 0;
}
